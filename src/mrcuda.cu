#include "miracl.cuh"

__FUNCTION_HEADER__ void *cuda_calloc(size_t nitems, size_t size)
{
    void *p;

    // 分配失败重复分配策略
    // do
    // {
    //     p = malloc(nitems * size);
    // } while (p == nullptr);
    // memset(p, 0, nitems * size);
    hipMalloc((void **)&p, nitems * size); // p = malloc(nitems * size);
    if (p)
        memset(p, 0, nitems * size);

    return p;
}

__FUNCTION_HEADER__ size_t cuda_strlen(const char *str)
{
    const char *s;

    for (s = str; *s; ++s)
        ;
    return (s - str);
}

__FUNCTION_HEADER__ int cuda_memcmp(const void *str1, const void *str2, size_t n)
{
    unsigned char *s1 = (unsigned char *)str1, *s2 = (unsigned char *)str2;
    if (!n)
        return 0;

    while (--n && *s1 && *s1 == *s2)
    {
        s1++;
        s2++;
    }

    return *(unsigned char *)s1 - *(unsigned char *)s2;
}

// __FUNCTION_HEADER__ int cuda_putchar(int ch)
// {
//     return printf("%c", ch);
// }

__FUNCTION_HEADER__ void cuda_exit(int status)
{
#ifndef _MY_NO_DEBUG_
    printf("exit(%d) !", status);
#endif
//    while (1)
//        ;
}